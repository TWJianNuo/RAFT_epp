#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include <hip/hip_math_constants.h>

namespace {

}

__global__ void epp_inflation_cuda_kernel(
    const torch::PackedTensorAccessor<int,4,torch::RestrictPtrTraits,size_t> instance,
    torch::PackedTensorAccessor<float,5,torch::RestrictPtrTraits,size_t> infdst,
    const torch::PackedTensorAccessor<float,4,torch::RestrictPtrTraits,size_t> infsrc,
    const int height,
    const int width,
    const int bs,
    const int infheight,
    const int infwidth
    ) {
    int m;
    int n;

    for(int i = threadIdx.x; i < height * width; i = i + blockDim.x){
        m = i / width;
        n = i - m * width;
        if (instance[blockIdx.x][0][m][n] > -1){
            for(int p1 = 0; p1 < infheight; p1++){
                for(int p2 = 0; p2 < infwidth; p2++){
                    infdst[blockIdx.x][m][n][p1][p2] = infsrc[blockIdx.x][instance[blockIdx.x][0][m][n]][p1][p2];
                }
            }
        }
    }
    return;

    }

__global__ void epp_compressio_cuda_kernel(
    const torch::PackedTensorAccessor<int,4,torch::RestrictPtrTraits,size_t> instance,
    torch::PackedTensorAccessor<float,4,torch::RestrictPtrTraits,size_t> compdst,
    torch::PackedTensorAccessor<float,5,torch::RestrictPtrTraits,size_t> compsrc,
    const int height,
    const int width,
    const int bs,
    const int compheight,
    const int compwidth
    ) {
    int m;
    int n;

    float inc = 0.1f;
    float tmpsum = 0;


    for(int i = threadIdx.x; i < height * width; i = i + blockDim.x){
        m = i / width;
        n = i - m * width;
        // if (instance[blockIdx.x][0][m][n] > -1){
            for(int p1 = 0; p1 < compheight; p1++){
                for(int p2 = 0; p2 < compwidth; p2++){
                    // atomicAdd((float*)&compdst[blockIdx.x][instance[blockIdx.x][0][m][n]][p1][p2], compsrc[blockIdx.x][m][n][p1][p2]);
                    // compsrc[blockIdx.x][m][n][p1][p2] = -1;
                    // atomicAdd((float*)&compdst[0][0][0][0], (float)compsrc[blockIdx.x][m][n][p1][p2]);
                    // atomicAdd((float*)&compdst[0][0][0][0], 1.0f);
                    // atomicAdd((float*)&compdst[blockIdx.x][instance[blockIdx.x][0][m][n]][p1][p2], 0.000002f);
                    // atomicAdd((float*)&compdst[0][0][0][0], inc);
                    atomicAdd((float*)&compdst[0][0][0][0], 0.1f);
                    atomicAdd((float*)&compdst[1][0][0][0], 1.0f);
                }
            }
        // }
    }
    return;

    }

void epp_inflation_cuda(
    torch::Tensor instance,
    torch::Tensor infdst,
    torch::Tensor infsrc,
    int height,
    int width,
    int bs,
    int infheight,
    int infwidth
    ){
      const int threads = 512;

      epp_inflation_cuda_kernel<<<bs, threads>>>(
            instance.packed_accessor<int,4,torch::RestrictPtrTraits,size_t>(),
            infdst.packed_accessor<float,5,torch::RestrictPtrTraits,size_t>(),
            infsrc.packed_accessor<float,4,torch::RestrictPtrTraits,size_t>(),
            height,
            width,
            bs,
            infheight,
            infwidth
            );
    return;
    }

void epp_compression_cuda(
    torch::Tensor instance,
    torch::Tensor compdst,
    torch::Tensor compsrc,
    int height,
    int width,
    int bs,
    int compheight,
    int compwidth
    ){
      const int threads = 512;

      epp_compressio_cuda_kernel<<<bs, threads>>>(
            instance.packed_accessor<int,4,torch::RestrictPtrTraits,size_t>(),
            compdst.packed_accessor<float,4,torch::RestrictPtrTraits,size_t>(),
            compsrc.packed_accessor<float,5,torch::RestrictPtrTraits,size_t>(),
            height,
            width,
            bs,
            compheight,
            compwidth
            );
    return;
    }